
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define START_LENGTH 128
#define MAX_BLOCK_SIZE 64

int predicate(int value) {
    if( (value > 7) && (value % 2) && !(value % 3) ) {
        return 1;
    } else {
        return 0;
    }
}

int* check_array(int *array, int length) {
    int *array_out = (int*)malloc(sizeof(int) * length);
    for(int i=0; i<length; i++) {
        array_out[i] = predicate(array[i]);
    }
    return array_out;
}

int* prefix_sum(int *array, int length) {
    int *array_out = (int*)malloc(sizeof(int) * length);
    array_out[0] = array[0];
    for(int i=1; i<length; i++) {
        array_out[i] = array[i] + array_out[i-1];
    }
    return array_out;
}

int* sieve_out(int *values, int *booleans, int *indexes, int *length) {
    int new_length = 0;
    for(int i=0; i<*length; i++) {
        if(booleans[i]) {
            new_length++;
        }
    }

    int *array_out = (int*)malloc(sizeof(int) * new_length);
    for(int i=0; i<*length; i++) {
        if(booleans[i]) {
            array_out[indexes[i]-1] = values[i];
        }
    }

    *length = new_length;
    return array_out;
}

void print_array(int *array, int length) {
    int *ptr = array;
    int counter = 0;
    while(ptr < array+length) {
        printf("array[%d] = %d\n", counter, *ptr);
        counter++;
        ptr++;
    }
}

int* seq_filter(int *array, int *length) {
    int *checked = check_array(array, *length);
    int *aggregated = prefix_sum(checked, *length);
    int *filtered = sieve_out(array, checked, aggregated, length);

    return filtered;
}

// #pragma OPENCL EXTENSION cl_khr_global_int32_base_atomics : enable

__global__ void filterKernel(int *values, int *bools, int *prefix_sum, int *output, int *array_length, int *semaphore) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // <---------<< 
    int covered_id = *array_length-1 - 2*idx;
    int threads = *array_length / 2;
    int blocks = threads / MAX_BLOCK_SIZE + (threads % MAX_BLOCK_SIZE > 0);
    int modulo = 2;

    // printf("Idx: %d, blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", idx, blockIdx.x, blockDim.x, threadIdx.x);

    if((values[covered_id] > 7) && (values[covered_id] % 2) && !(values[covered_id] % 3)) {
        bools[covered_id] = 1;
        prefix_sum[covered_id] = 1;
    } else {
        bools[covered_id] = 0;
        prefix_sum[covered_id] = 0;
    }

    if(covered_id-1 >= 0) {
        if((values[covered_id-1] > 7) && (values[covered_id-1] % 2) && !(values[covered_id-1] % 3)) {
            bools[covered_id-1] = 1;
            prefix_sum[covered_id-1] = 1;
        } else {
            bools[covered_id-1] = 0;
            prefix_sum[covered_id-1] = 0;
        }
    }

    // //synchronizing before prefix_sum;
    __syncthreads();  // <---------<< 
    // opencl --> int atomic_inc(volatile __global *int)
    // if(threads > MAX_BLOCK_SIZE) {
    //     if(threadIdx.x == 0)
    //         atomicInc((unsigned int*)semaphore, *array_length);
    //     while(*semaphore != blocks) {}
    //     if(idx == 0)
    //         atomicXor(semaphore, *semaphore);
    // }

    // counting the prefix sum
    while(modulo < (2**array_length)) {
        // int active_threads = (threads + (modulo/2 -1)) / (modulo/2);
        if((covered_id % modulo) == ((*array_length-1) % modulo)) {
            if((covered_id - modulo/2) >= 0) {
                prefix_sum[covered_id] += prefix_sum[covered_id-modulo/2];
            }

            // sync in between levels of summing
            __syncthreads();  // <---------<< 
        }
        modulo *= 2;
    }

    if(covered_id == *array_length-1) {
        prefix_sum[covered_id] = 0;
    }

    // sync after up-sweep
    __syncthreads();  // <---------<< 

    while(modulo >= 2) {
        if((covered_id % modulo) == ((*array_length-1) % modulo)) {
            if((covered_id - modulo/2) >= 0) {
                int temp = prefix_sum[covered_id];
                prefix_sum[covered_id] += prefix_sum[covered_id - modulo/2];
                prefix_sum[covered_id - modulo/2] = temp;
            }
        }

        __syncthreads();  // <---------<< 
        __syncthreads();  // <---------<< 
        modulo /= 2;
    }

    if(bools[covered_id])
        prefix_sum[covered_id] += 1;
    if((covered_id-1) >= 0)
        if(bools[covered_id-1])
            prefix_sum[covered_id-1] += 1;

    __syncthreads();  // <---------<< 

    if(bools[covered_id]) {
        output[prefix_sum[covered_id]-1] = values[covered_id];
    }
    if(((covered_id - 1) >= 0) && (bools[covered_id-1])) {
        output[prefix_sum[covered_id-1]-1] = values[covered_id-1];
    }

    if(idx == 0) {
        *array_length = prefix_sum[*array_length-1];
    }

}

int* gpu_filter(int* array, int *length) {
    int *values, *bools, *prefix_sum, *output, *array_length, *semaphore;
    int size = sizeof(int) * *length;
    hipMalloc(&values, size);  // <---------<< 
    hipMalloc(&bools, size);  // <---------<< 
    hipMalloc(&prefix_sum, size);  // <---------<< 
    hipMalloc(&output, size);  // <---------<< 
    hipMalloc(&array_length, sizeof(int));  // <---------<< 
    hipMalloc(&semaphore, sizeof(int)*2);  // <---------<< 
    hipMemcpy(values, array, size, hipMemcpyHostToDevice);  // <---------<< 
    hipMemcpy(array_length, length, sizeof(int), hipMemcpyHostToDevice);  // <---------<< 

    
    int threads = *length / 2;
    int blocks = threads / MAX_BLOCK_SIZE + (threads % MAX_BLOCK_SIZE > 0);
    int threads_in_block = (threads + blocks-1) / blocks;

    filterKernel<<<blocks, threads_in_block>>>(values, bools, prefix_sum, output, array_length, semaphore);  // <---------<< 

    hipMemcpy(length, array_length, sizeof(int), hipMemcpyDeviceToHost);  // <---------<< 
    int new_size = *length * sizeof(int);  // <---------<< 
    hipMemcpy(array, output, new_size, hipMemcpyDeviceToHost);  // <---------<< 
    hipFree(&values);  // <---------<< 
    hipFree(&bools);  // <---------<< 
    hipFree(&prefix_sum);  // <---------<< 
    hipFree(&output);  // <---------<< 
    hipFree(&array_length);  // <---------<< 
    hipFree(&semaphore);  // <---------<< 

    return array;
}

int main() {
    srand(time(NULL));
    int length = START_LENGTH;

    int *array = (int*)malloc(sizeof(int) * length);
    for(int i=0, *ptr=array; i<length; i++, ptr++) {
        *ptr = rand()%length;
    }

    int* filtered = seq_filter(array, &length);
    print_array(filtered, length);

    printf("\n-------------------- array[0] = %d --------------------\n\n", array[0]);

    length = START_LENGTH;
    int* gpu_filtered = gpu_filter(array, &length);
    print_array(gpu_filtered, length);

    bool coherent = true;
    int iter = 0;
    while(coherent && (iter < length)) {
        if(filtered[iter] != array[iter]) {
            coherent = false;
        }
        iter++;
    }
    if(coherent)
        printf("\x1b[1;40;32mCorrect :)\x1b[0m\n");
    else
        printf("\x1b[1;40;31mIncorrect!\x1b[0m\n");
}